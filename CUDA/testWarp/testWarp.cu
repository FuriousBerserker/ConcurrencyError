
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void shiftArray(unsigned* a, unsigned size) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    while (tid < size) {
        a[tid] = a[(tid + 1) % size];
        tid += blockDim.x;
    }
}

int main() {
    unsigned size = 100, i;
    unsigned size_in_byte = size * sizeof(unsigned);
    unsigned *array = (unsigned*)malloc(size_in_byte);
    unsigned *d_array;
    for (i = 0; i < size; i++) {
        array[i] = i;    
    }
    hipMalloc(&d_array, size_in_byte);
    hipMemcpy(d_array, array, size_in_byte, hipMemcpyHostToDevice);

    shiftArray<<<1, 32>>> (d_array, size);

    hipMemcpy(array, d_array, size_in_byte, hipMemcpyDeviceToHost);

    for (i = 0; i < size; i++) {
        printf("array[%d] = %d\n", i, array[i]);
    }
    return 0;
}
