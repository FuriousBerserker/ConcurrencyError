
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaErrorCheck(call)                                         \
    do {                                                             \
        hipError_t cuErr = call;                                    \
        if (hipSuccess != cuErr) {                                  \
            printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, \
                   hipGetErrorString(cuErr));                       \
            exit(0);                                                 \
        }                                                            \
    } while (0)



#define BLOCK_NUM 100
#define BLOCK_DIM 512
__device__ int count = 0;

__global__ void getThreadNum() {
    int oldVal, returnVal;
    do {
        oldVal = count;
        returnVal = atomicCAS(&count, oldVal, oldVal + 1);
    } while (oldVal != returnVal);
}

int main() {
    int result = -1;
    getThreadNum<<<BLOCK_NUM, BLOCK_DIM>>>();
    cudaErrorCheck(hipGetLastError());
    hipDeviceSynchronize();
    int *count_addr;
    cudaErrorCheck(hipGetSymbolAddress((void**)&count_addr, HIP_SYMBOL(count)));
    cudaErrorCheck(hipMemcpy(&result, count_addr, sizeof(int), hipMemcpyDeviceToHost)); 
    printf("thread num, expected = %d, actual = %d\n", BLOCK_NUM * BLOCK_DIM, result);
    return 0;
}


